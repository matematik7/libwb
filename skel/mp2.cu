#include "hip/hip_runtime.h"
#include    <wb.h>

#define wbCheck(stmt) do {                                 \
        hipError_t err = stmt;                            \
        if (err != hipSuccess) {                          \
            wbLog(ERROR, "Failed to run stmt ", #stmt);    \
            return -1;                                     \
        }                                                  \
    } while(0)

// compute matrix size in ram from dimensions
#define MATRIX_SIZE(a,b) (a) * (b) * sizeof(float)

// Compute C = A * B
__global__ void matrixMultiply(float * A, float * B, float * C,
			       int numARows, int numAColumns,
			       int numBRows, int numBColumns,
			       int numCRows, int numCColumns) {
    //@@ Insert code to implement matrix multiplication here
}

int main(int argc, char ** argv) {
    wbArg_t args;
    float * hostA; // The A matrix
    float * hostB; // The B matrix
    float * hostC; // The output C matrix
    float * deviceA;
    float * deviceB;
    float * deviceC;
    int numARows; // number of rows in the matrix A
    int numAColumns; // number of columns in the matrix A
    int numBRows; // number of rows in the matrix B
    int numBColumns; // number of columns in the matrix B
    int numCRows; // number of rows in the matrix C (you have to set this)
    int numCColumns; // number of columns in the matrix C (you have to set this)

    args = wbArg_read(argc, argv);

    wbTime_start(Generic, "Importing data and creating memory on host");
    hostA = (float *) wbImport(wbArg_getInputFile(args, 0), &numARows, &numAColumns);
    hostB = (float *) wbImport(wbArg_getInputFile(args, 1), &numBRows, &numBColumns);
    //@@ Set numCRows and numCColumns
    numCRows = numARows;
    numCColumns = numBColumns;
    //@@ Allocate the hostC matrix
    hostC = (float *) malloc(MATRIX_SIZE(numCRows, numCColumns));
    wbTime_stop(Generic, "Importing data and creating memory on host");

	// cpu computation
	for (int i = 0; i < numCRows; i++) {
		for (int j = 0; j < numCColumns; j++) {
			float tmp = 0;
			for (int n = 0; n < numAColumns; n++) {
				tmp += hostA[i*numAColumns + n] * hostB[j + n*numBColumns];
			}
			hostC[i * numCColumns + j] = tmp;
		}
	}

    wbLog(TRACE, "The dimensions of A are ", numARows, " x ", numAColumns);
    wbLog(TRACE, "The dimensions of B are ", numBRows, " x ", numBColumns);

    wbTime_start(GPU, "Allocating GPU memory.");
    //@@ Allocate GPU memory here

    wbTime_stop(GPU, "Allocating GPU memory.");

    wbTime_start(GPU, "Copying input memory to the GPU.");
    //@@ Copy memory to the GPU here

    wbTime_stop(GPU, "Copying input memory to the GPU.");
    
    //@@ Initialize the grid and block dimensions here
    
    wbTime_start(Compute, "Performing CUDA computation");
    //@@ Launch the GPU Kernel here

    hipDeviceSynchronize();
    wbTime_stop(Compute, "Performing CUDA computation");
    
    wbTime_start(Copy, "Copying output memory to the CPU");
    //@@ Copy the GPU memory back to the CPU here

    wbTime_stop(Copy, "Copying output memory to the CPU");

    wbTime_start(GPU, "Freeing GPU Memory");
    //@@ Free the GPU memory here

    wbTime_stop(GPU, "Freeing GPU Memory");

    wbSolution(args, hostC, numCRows, numCColumns);

    free(hostA);
    free(hostB);
    free(hostC);

    return 0;
}

